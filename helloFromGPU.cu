
#include <hip/hip_runtime.h>
#include <stdio.h>

//__global__修飾子は，この関数がCPUから呼び出され，GPUで実行されることをコンパイラに認識させる．
__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");
    //<<<...>>>はホストスレッドからのデバイスコードの呼び出しを指定する，
    //カーネルは一連のスレッドによって実行され，すべてのスレッドが同じコードを実行する．
    //<<<...>>>で囲まれているパラメータは，このカー熱を実行するスレッドの数を指定する．
    //この例では，GPUスレッドを10個実行することになる．
    helloFromGPU <<<1, 10>>>();
    hipDeviceReset();
    return 0;
}